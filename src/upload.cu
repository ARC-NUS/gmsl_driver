
#include <hip/hip_runtime.h>
// memcpy from gpu to cpu

void cudaCopy(unsigned char* dst, unsigned char* src, int bytes)
{
	hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost);
	return;
}
