#include "hip/hip_runtime.h"
// to upload the code from GPu to CPU

#include <stdint.h>
#include <iostream>
//headers for image msg
//#include <sensor_msgs/Image.h>
//#include <sensor_msgs/image_encodings.h>

//set static global var texture. texture cannot be passed as argument !! check if param is correct
texture<float, hipTextureType2D, hipReadModeElementType> textureRef;


	__global__
void copyKernel(unsigned char* d_array, int width, int height, float theta)
{
	// Calculate normalized texture coordinates !! check if correct/ what it means
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;

	float v = y / (float)height;

	// Transform coordinates
	u -= 0.5f;

	v -= 0.5f; 

	float tu = u * cosf(theta) - v * sinf(theta) + 0.5f ;

	float tv = v * cosf(theta) + u * sinf(theta) + 0.5f ;

	// Read from texture and write to global memory
	d_array[y * width + x] = tex2D(textureRef, tu, tv);
//	d_array[y * width + x] = 100;
	return;
}

__global__ void fakeArray(unsigned char* d_array, int width, int height, float theta)
{
	for(int i = 0 ; i <width; ++i)
		for(int j = 0 ; j < height; ++j)
		{
			d_array[i*width + j] = 10;
		}
	return;
}


void cuda2sharedMem(unsigned char* d_array, hipArray_t c_array, int width, int height)
{
	//hipSetDevice(1);  // to use shared memory space

	// ---- convert to texture ---- //

	//set texture param !! check to see what does these param mean
	textureRef.addressMode[0] = hipAddressModeWrap;
	textureRef.addressMode[1] = hipAddressModeWrap;
	textureRef.filterMode = hipFilterModeLinear;
	textureRef.normalized = true;

	//set cuda array channel descp !! look into how to get this from the cuda array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

	hipBindTextureToArray(textureRef, c_array, channelDesc); 


	// ---- copy to an array on the GPU ---- //

	// unsigned char* d_array;
	// int size = width * height * sizeof(uint8_t);
	// hipMalloc(&d_array, size);

	dim3 blockSize(16, 16);
	dim3 numBlocks( (width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y );

	// std::cerr << " copying with " << numBlocks.x + numBlocks.y << " of blocks, of size " << blockSize.x  << " by " << blockSize.y << "each\n";
	// std::cerr << "copying the image of width: " << width;
	//call kernel to copy
	copyKernel<<<numBlocks,blockSize>>>(d_array, width, height, 90.0);

//	fakeArray<<<1,1>>>(d_array, width, height, 90.0);
//	std::cerr << "cuda generted fake img\n"; 
	hipDeviceSynchronize();

	// ---- unbind n release resources ---- //
	//unbind to texture? !! ?
	hipUnbindTexture(textureRef);

	return;
}


void cudaCopy(unsigned char* dst, unsigned char* src, int bytes)
{
	hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost);
	//	hipMemcpy(dst, src, bytes, hipMemcpyHostToHost);
	return;
}


void cuda2HostArray(hipArray_t c_array, unsigned char* h_array, int width, int height, int bytes)
{
	unsigned char* d_array;
	int size = bytes; //width * height * sizeof(uint8_t);
	hipError_t code;
	code = hipMalloc(&d_array, size);
	if(code != hipSuccess)
	{
		std::cerr << "cuda malloc failed\n";
	}

	// d_array = cuda2sharedMem(d_array, c_array, width, height);
	cuda2sharedMem(d_array, c_array, width, height);

	// hipMemcpy(h_array, * c_array, bytes, hipMemcpyDeviceToHost);
	// hipMemcpy(h_array, d_array, bytes, hipMemcpyHostToHost);
	hipMemcpy(h_array, d_array, bytes, hipMemcpyDeviceToHost);

	hipFree(d_array);

	std::cerr << "returned resources, leaving cuda\n";

	return;
}


void cudaMemCpyArr(hipArray_t  c_array, unsigned char* h_array, int width, int height, int bytes)
// void cudaMemCpyArr(const struct hipArray * c_array, unsigned char* h_array, int width, int height, int bytes)
{
/**
hipError_t hipMemcpy2DFromArray 	( 	void *  	dst,
		size_t  	dpitch,
		const struct hipArray *  	src,
		size_t  	wOffset,
		size_t  	hOffset,
		size_t  	width,
		size_t  	height,
		enum hipMemcpyKind  	kind	 
	) 	
**/	
/**	hipError_t code; 	
	code = hipMemcpy2DFromArray(h_array, width, c_array, 0, 0, width, height, hipMemcpyDeviceToHost);
	if(code != hipSuccess)
	{
		std::cerr << "cuda memcpy array failed: " << hipGetErrorString(code) << std::endl;
	}
**/

/**	hipError_t hipMemcpyFromArray 	( 	void *  	dst,
		const struct hipArray *  	src,
		size_t  	wOffset,
		size_t  	hOffset,
		size_t  	count,
		enum hipMemcpyKind  	kind	 
	) 	
**/

	float *testArray = new float[width * height * 4];	
	// uint8_t *testArray = new uint8_t[width * height * 4];	
	hipArray* cuArray;
	hipChannelFormatDesc channelD = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMallocArray(&cuArray, &channelD, width*height*4, 0); 
	hipError_t code;       
        // code = hipMemcpyFromArray(h_array, c_array, 0, 0, bytes, hipMemcpyDeviceToHost);
        code = hipMemcpyFromArray(testArray, cuArray, 0, 0, 1, hipMemcpyDeviceToHost);
        if(code != hipSuccess)
        {
                std::cerr << "cuda memcpy array failed: " << hipGetErrorString(code) << std::endl;
        }

	hipDeviceSynchronize();

	return;
}
